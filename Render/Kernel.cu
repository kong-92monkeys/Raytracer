#include "hip/hip_runtime.h"
#include "Kernel.h"
#include "PixelHandler.h"
#include <>

namespace Render
{
	namespace Kernel
	{
		__global__ void launch_device(
			ResourceContext const resourceContext,
			SurfaceContext const surfaceContext)
		{
			auto const gidX{ (blockIdx.x * blockDim.x) + threadIdx.x };
			auto const gidY{ (blockIdx.y * blockDim.y) + threadIdx.y };

			PixelHandler pixelHandler{ gidX, gidY, surfaceContext };

			if (!(pixelHandler.isValid()))
				return;

			uchar4 color{ 255, 0, 255, 255 };
			pixelHandler.set(color);
		}

		void launch(
			ResourceContext const &resourceContext,
			SurfaceContext const &surfaceContext,
			LaunchContext const &launchContext)
		{
			launch_device<<<
				launchContext.gridDim,
				launchContext.blockDim,
				0U,
				launchContext.stream>>>
				(resourceContext, surfaceContext);
		}
	}
}