#include "hip/hip_runtime.h"
#include "Kernel.h"
#include "PixelHandler.h"
#include <>

namespace Render
{
	namespace Kernel
	{
		__global__ void launch_device(
			ResourceContext const resourceContext,
			SurfaceContext const surfaceContext)
		{
			auto const gidX{ (blockIdx.x * blockDim.x) + threadIdx.x };
			auto const gidY{ (blockIdx.y * blockDim.y) + threadIdx.y };

			PixelHandler pixelHandler{ gidX, gidY, surfaceContext };

			if (!(pixelHandler.isValid()))
				return;

			float4 color{ 0.0f, 0.0f, 0.0f, 1.0f };

			float const rayX{ gidX * 1.0f };
			float const rayY{ gidY * 1.0f };

			float const deltaX{ rayX - resourceContext.sphereCenter.x };
			float const deltaY{ rayY - resourceContext.sphereCenter.y };
			float const radius{ resourceContext.sphereRadius };

			if ((radius * radius) > ((deltaX * deltaX) + (deltaY * deltaY)))
				color.x = 1.0f;

			pixelHandler.set(color);
		}

		void launch(
			ResourceContext const &resourceContext,
			SurfaceContext const &surfaceContext,
			LaunchContext const &launchContext)
		{
			launch_device<<<
				launchContext.gridDim,
				launchContext.blockDim,
				0U,
				launchContext.stream>>>
				(resourceContext, surfaceContext);
		}
	}
}