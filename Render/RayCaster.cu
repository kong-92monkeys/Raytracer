#include "hip/hip_runtime.h"
#include "RayCaster.h"
#include "CudaMath.h"

namespace Render
{
	namespace Kernel
	{
		__host__ RayCaster::RayCaster(
			Viewport const &viewport,
			uint32_t const surfaceWidth,
			uint32_t const surfaceHeight) noexcept :
			__viewport	{ viewport },
			__stepX		{ viewport.width / surfaceWidth },
			__stepY		{ viewport.height / surfaceHeight }
		{}

		__device__ Ray RayCaster::cast(
			uint32_t const x,
			uint32_t const y) const noexcept
		{
			float3 target{ __viewport.origin };
			target += (__viewport.right * (__stepX * x));
			target += (__viewport.down * (__stepY * y));

			auto const dir{ normalize(target - __viewport.eye) };
			return { __viewport.eye, dir };
		}
	}
}